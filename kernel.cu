#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void forwardStepKernel(double* c, double* a, double* b, double* d, int a_size, 
	int a_size_next, int c_size, int a_start_id, int c_start_id, int k)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < a_size_next)
	{
		int id_prev = a_start_id + idx * 2 + 1;       // предыдущие коэффициенты
		int id_prev_с = c_start_id + idx * 2 + 1;     //
		int id_cur = a_start_id + a_size + idx;   // текущие коэффициенты
		int id_cur_c = c_start_id + c_size + idx; //

		double repeat_coef_1 = c[id_prev_с - 1] / a[id_prev - 1];
		double repeat_coef_2 = b[id_prev_с] / a[id_prev + 1];
		if (k == 0)
		{
			if (idx > 0) c[id_cur_c - 1] = -repeat_coef_1 * c[id_prev_с - 2];

			a[id_cur] = a[id_prev] - repeat_coef_1 * b[id_prev_с - 1] - repeat_coef_2 * c[id_prev_с];

			if (idx < a_size_next - 1) b[id_cur_c] = -repeat_coef_2 * b[id_prev_с + 1];
		}
		d[id_cur] = d[id_prev] - repeat_coef_1 * d[id_prev - 1] - repeat_coef_2 * d[id_prev + 1];
	}
}

__global__ void backStepKernel(double* c, double* a, double* b, double* d, int a_size, 
	int c_size, int a_start_id, int c_start_id, int a_sum, int i, int q_input, int eq_use_count, int I)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < q_input - 1 && idx < eq_use_count)
	{
		int shift = 1 << i;
		int id_cur = a_sum + shift + idx * (shift << 1) - 1;
		int j_2 = idx * 2;
		if (idx > 0 && idx < eq_use_count - 1)
		{
			d[id_cur] = (d[a_start_id + j_2] - c[c_start_id + j_2 - 1] * d[id_cur - shift] -
				b[c_start_id + j_2] * d[id_cur + shift]) / a[a_start_id + j_2];
		}
		else if (idx == 0)
		{
			d[id_cur] = (d[a_start_id] - b[c_start_id] * d[id_cur + shift]) / a[a_start_id];
		}
		else
		{
			d[id_cur] = (d[a_start_id + a_size - 1] - c[c_start_id + c_size - 1] * d[id_cur - shift]) 
				/ a[a_start_id + a_size - 1];
		}
	}
	else if (idx == I / 2) d[a_sum + I / 2] = d[a_sum - 1] / a[a_sum - 1];
}

__global__ void swapDArrElem(double* d, int I, int a_sum)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < I)
	{
		d[idx] = d[a_sum + idx];
	}
}

void print_jagged_array(double* arr, int* sizes, int* start_ids, int q)
{
	for (int i = 0; i < q; ++i)
	{
		for (int j = start_ids[i]; j < start_ids[i] + sizes[i]; ++j)
		{
			cout << arr[j] << " ";
		}
		cout << endl;
	}
}

int main()
{
	// моделирование распространения тепла в однородном, теплоизолированном с боков, шаре
	double k_input = 0.59; // коэффициент теплопроводности
	double c_input = 1.65; // объёмная теплоёмкость
	double a_input = 0.6; // коэффициент температуропроводности
	double R_input = 6; // радиус шара

	double duration_input = 50; // время моделирования
	int q_input = 15; // двойка в степени для кол-ва уравнений
	int I = (1 << q_input) - 1; // кол-во уравнений – двойка в степени минус один
	int K = 5000; // кол-во временных слоёв

	bool is_print_values = false; // выводить ли вычисленные значения

	cout << "I = " << I << ", K = " << K << ", I * K = " << I * K << endl;

	double h_r = R_input / I;
	double h_t = duration_input / K;

	double* r_linspace = new double[I];
	for (int i = 0; i < I; ++i) {
		r_linspace[i] = i * R_input / (I - 1);
	}

	double* t_linspace = new double[K];
	for (int k = 0; k < K; ++k) {
		t_linspace[k] = k * duration_input / (K - 1);
	}

	double gamma = 6 * k_input / c_input * h_t / (h_r * h_r);
	double xi = 2 * k_input / c_input * h_t / h_r;
	double eta = k_input / c_input * h_t / (h_r * h_r);

	int* c_sizes = new int[q_input - 1];
	int* a_sizes = new int[q_input];

	int* c_start_ids = new int[q_input - 1];
	int* a_start_ids = new int[q_input];

	c_start_ids[0] = 0;
	a_start_ids[0] = 0;
	for (int i = 0; i < q_input; ++i)
	{
		int I_reduce = I >> i;

		if (i < q_input - 1) c_sizes[i] = I_reduce - 1;
		a_sizes[i] = I_reduce;
		if (i > 0)
		{
			if (i < q_input - 1) c_start_ids[i] = c_start_ids[i - 1] + c_sizes[i - 1];
			a_start_ids[i] = a_start_ids[i - 1] + a_sizes[i - 1];
		}
	}

	int c_sum = 0;
	int a_sum = 0;
	for (int i = 0; i < q_input; ++i)
	{
		if (i < q_input - 1) c_sum += c_sizes[i];
		a_sum += a_sizes[i];
	}

	// выделение памяти на CPU 
	double* c = new double[c_sum]; // нижняя диагональ
	double* a = new double[a_sum]; // главная диагональ
	double* b = new double[c_sum]; // верхняя диагональ
	double* d = new double[a_sum + I]; // правая часть (искомые неизвестные и начальное условие)

	// выделение памяти на GPU
	double* d_c; // нижняя диагональ
	double* d_a; // главная диагональ
	double* d_b; // верхняя диагональ
	double* d_d; // правая часть (искомые неизвестные и начальное условие)
	hipMalloc(&d_c, c_sum * sizeof(double));
	hipMalloc(&d_a, a_sum * sizeof(double));
	hipMalloc(&d_b, c_sum * sizeof(double));
	hipMalloc(&d_d, (a_sum + I) * sizeof(double));

	for (int i = 0; i < I; ++i) { // инициализация массивов
		if (i > 0 && i < I - 1)
		{
			c[i - 1] = -eta;
			a[i] = 1 + xi / r_linspace[i] + 2 * eta;
			b[i] = -(xi / r_linspace[i] + eta);
		}
		else if (i == 0)
		{
			a[0] = 1 + gamma;
			b[0] = -gamma;
		}
		else
		{
			c[I - 2] = -2 * eta - xi / R_input;
			a[I - 1] = 1 + xi / R_input + 2 * eta;
		}
		d[i] = 12 * exp(-pow((r_linspace[i] / a_input), 2));
	}

	// копируем данные на GPU
	hipMemcpy(d_c, c, c_sum * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_a, a, a_sum * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, c_sum * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_d, d, (a_sum + I) * sizeof(double), hipMemcpyHostToDevice);

	ofstream file("dArray.txt");
	if (!file.is_open()) {
		cerr << "Error!\n";
		return 1;
	}
	for (int k = 0; k < K - 1; ++k) // суслик
	{
		int blockSize = 256;
		int gridSize = (I + blockSize - 1) / blockSize;

		hipEvent_t start, stop;
		if (k == 0)
		{
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start);
		}

		for (int i = 0; i < q_input - 1; ++i) {
			forwardStepKernel << <gridSize, blockSize >> > (d_c, d_a, d_b, d_d,
				a_sizes[i], a_sizes[i + 1], c_sizes[i], a_start_ids[i], c_start_ids[i], k);
		}
		for (int i = q_input - 1, eq_use_count = 1; i >= 0; --i, eq_use_count <<= 1) {
			backStepKernel << <gridSize, blockSize >> > (d_c, d_a, d_b, d_d,
				a_sizes[i], c_sizes[i], a_start_ids[i], c_start_ids[i], a_sum, i, q_input, eq_use_count, I);
		}

		if (k == 0)
		{
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float total_time_ms;
			hipEventElapsedTime(&total_time_ms, start, stop);
			cout << "Total execution time: " << total_time_ms << " ms\n";
			hipEventDestroy(start);
			hipEventDestroy(stop);
		}

		hipMemcpy(d, d_d, (a_sum + I) * sizeof(double), hipMemcpyDeviceToHost);

		if (k < K - 1)
		{
			swapDArrElem << < gridSize, blockSize >> > (d_d, I, a_sum);
		}
		hipDeviceSynchronize();

		if (k == 0)
		{
			if (is_print_values)
			{
				cout << "c_array:\n"; // вывод нижней диагонали
				print_jagged_array(c, c_sizes, c_start_ids, q_input - 1);
				cout << "a_array:\n"; // вывод главной диагонали
				print_jagged_array(a, a_sizes, a_start_ids, q_input);
				cout << "b_array:\n"; // вывод верхней диагонали
				print_jagged_array(b, c_sizes, c_start_ids, q_input - 1);
				cout << "d_array:\n"; // вывод правой части
			}

			file << R_input << " " << I << " " << duration_input << " " << K << endl;
			for (int i = 0; i < I; ++i) file << d[i] << " ";
			file << endl;
		}
		if (is_print_values)
		{
			cout << "k = " << k << endl;
			print_jagged_array(d, a_sizes, a_start_ids, q_input);
		}

		for (int i = 0; i < I; ++i) file << d[a_sum + i] << " ";
		file << endl;
	}
	if (is_print_values)
	{
		cout << "k = " << K - 1 << endl;
		for (int i = 0; i < I; ++i) cout << d[a_sum + i] << " "; // вывод последнего слоя K
	}

	system("python graph.py");

	delete[] r_linspace;
	delete[] t_linspace;
	delete[] c;
	delete[] a;
	delete[] b;
	delete[] d;
	hipFree(d_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_d);
	delete[] c_sizes;
	delete[] a_sizes;
	delete[] c_start_ids;
	delete[] a_start_ids;
}